#include "hip/hip_runtime.h"
#include <vector>
#include <cnpy.h>
#include <hip/hip_runtime.h>
#include <limits>
// CUDA ядро для вычисления расстояний и нахождения ближайших соседей
__global__ void find_nearest_neighbors(const float* query, const float* database, int* indices, float* distances, int num_db, int dim, int num_neighbors) {
    int db_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (db_idx < num_db) {
        float dist = 0.0f;
        for (int i = 0; i < dim; ++i) {
            float diff = query[i] - database[db_idx * dim + i];
            dist += diff * diff;
        }
        dist = sqrtf(dist);

        // Update nearest neighbors
        for (int k = 0; k < num_neighbors; ++k) {
            if (dist < distances[k] || distances[k] < 0) {
                // Shift the rest
                for (int l = num_neighbors - 1; l > k; --l) {
                    distances[l] = distances[l - 1];
                    indices[l] = indices[l - 1];
                }
                // Insert the new distance
                distances[k] = dist;
                indices[k] = db_idx;
                break;
            }
        }
    }
}


// Функция для загрузки данных из .npy файла и передачи на GPU
void load_data_to_gpu(const std::string& filename, float** d_data, size_t num_elements) {
    // Загрузка данных из .npy файла
    cnpy::NpyArray arr = cnpy::npy_load(filename);
    float* h_data = arr.data<float>();

    // Выделение памяти на GPU
    hipMalloc(d_data, num_elements * sizeof(float));
    // Копирование данных с CPU на GPU
    hipMemcpy(*d_data, h_data, num_elements * sizeof(float), hipMemcpyHostToDevice);
}

// Основная функция для выполнения поиска ближайших соседей
void find_nearest_neighbors_gpu(const std::string& query_file, const std::string& database_file, std::vector<int>& indices, std::vector<float>& distances, int num_db, int dim, int num_neighbors) {
    float* d_query;
    float* d_database;
    int* d_indices;
    float* d_distances;

    size_t query_size = dim;
    size_t database_size = num_db * dim;
    size_t indices_size = num_neighbors * sizeof(int);
    size_t distances_size = num_neighbors * sizeof(float);

    // Загрузка данных на GPU
    load_data_to_gpu(query_file, &d_query, query_size);
    load_data_to_gpu(database_file, &d_database, database_size);

    // Выделение памяти для результатов на GPU
    hipMalloc(&d_indices, indices_size);
    hipMalloc(&d_distances, distances_size);

    // Инициализация расстояний на GPU значением инфинити
    std::vector<float> init_distances(num_neighbors, std::numeric_limits<float>::infinity());
    hipMemcpy(d_distances, init_distances.data(), distances_size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (num_db + block_size - 1) / block_size;
    find_nearest_neighbors<<<grid_size, block_size>>>(d_query, d_database, d_indices, d_distances, num_db, dim, num_neighbors);

    // Проверка на ошибки в запуске ядра
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    // Копирование результатов обратно на CPU
    hipMemcpy(indices.data(), d_indices, indices_size, hipMemcpyDeviceToHost);
    hipMemcpy(distances.data(), d_distances, distances_size, hipMemcpyDeviceToHost);

    // Освобождение памяти на GPU
    hipFree(d_query);
    hipFree(d_database);
    hipFree(d_indices);
    hipFree(d_distances);
}

int main() {
    const int d = 1024;  // Размерность векторов
    const int nb = 27000; // Количество векторов в базе данных
    const int k = 4;     // Количество ближайших соседей

    std::vector<int> indices(k);
    std::vector<float> distances(k);

    // Поиск ближайших соседей
    find_nearest_neighbors_gpu("../query.npy", "../vecs.npy", indices, distances, nb, d, k);

    // Вывод результатов (индексы и расстояния ближайших соседей)
    std::cout << "Nearest neighbors:\n";
    for (int j = 0; j < k; ++j) {
        std::cout << "Index: " << indices[j] << ", Distance: " << distances[j] << "\n";
    }

    return 0;
}